#include"header.h"

void call_kernel(vector <vector<int> >&fan,thrust::device_vector<int> &d_fan, vertex_buffer* &h_vb, vertex_buffer* &output, int row_size,int v_count)
{

  hipError_t err;
  vertex_buffer *d_vb;
  int *d_fan_ptr;
  vertex_buffer *d_result;
  int height;
  float *jacobian;
  function_info *funct;
  float *d1;



  d_fan_ptr = thrust::raw_pointer_cast(&d_fan[0]);


  err=hipMalloc((void**)&d_vb,sizeof(struct vertex_buffer)*v_count);
  if(err!=hipSuccess){
    cout<<"failed to allocate memory for d_vb\n";
    exit(0);
  }



  err=hipMemcpy(d_vb,h_vb,sizeof(vertex_buffer)*v_count,hipMemcpyHostToDevice);
  if(err!=hipSuccess){
    cout<<"unable to copy host vertices lists to device vertices lists\n";
    exit(0);
  }


  output=new vertex_buffer[v_count];
  err=hipMalloc((void**)&d_result,sizeof(vertex_buffer)*v_count);
  if(err!=hipSuccess){
    cout<<"unable to allocate memory for d_result\n";
    exit(0);
  }

 

  err=hipMalloc((void**)&jacobian,sizeof(float)*2*v_count*row_size);
  if(err!=hipSuccess){
    cout<<"unable to allocate memory for jacobain\n";
    exit(0);
  }

  err=hipMalloc((void**)&funct,sizeof(function_info*)*v_count*row_size);
  if(err!=hipSuccess){
    cout<<"unable to allocate memory to store function information\n";
    exit(0);
  }


  err=hipMalloc((void**)&d1,sizeof(float)*2*v_count*row_size);
  if(err!=hipSuccess){
    cout<<"unable to allocate memory to store the d information\n";
    exit(0);
   }


  
      //height=1024/row_size;
        height=(v_count-1)/512+1;

  //dim3 dimGrid(1,((v_count-1)/height+1),1);
  //dim3 dimBlock(row_size,height,1);

   dim3 dimGrid(1,height,1);
   dim3 dimBlock(1,512,1);


  hipEvent_t event1, event2;
  hipEventCreate(&event1);
  hipEventCreate(&event2);
  hipEventRecord(event1,0);
  new_angle_based<<<dimGrid,dimBlock>>>(d_vb,d_result,d_fan_ptr,v_count,row_size,jacobian,funct,d1);
  hipEventRecord(event2, 0);


  hipEventSynchronize(event2); //wait for the event to be executed!


  float dt_ms;
  hipEventElapsedTime(&dt_ms, event1, event2);

  printf("elapsed_time = %f\n",dt_ms);


  hipMemcpy(output,d_result,sizeof(vertex_buffer)*v_count,hipMemcpyDeviceToHost);


//  for(int i=0;i<v_count;i++)
//   cout<<output[i].x<<" "<<output[i].y<<endl;
}
