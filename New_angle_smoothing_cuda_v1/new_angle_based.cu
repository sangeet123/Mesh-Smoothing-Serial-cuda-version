#include "hip/hip_runtime.h"
#include"header.h"



void __global__ new_angle_based(const struct vertex_buffer *d_vb, struct vertex_buffer *result,int *fan,int vertices,int max,float *jacobian,function_info*funct,float*d1)
{
//__shared__ int fan_list[512][20];

//int tx=threadIdx.x;
int ty=threadIdx.y;
int count;
//int column=blockIdx.x * blockDim.x+tx;
int row=blockIdx.y *blockDim.y+ty;
struct vertex_buffer v0;
struct vertex_buffer v1;
struct vertex_buffer v2;
struct vertex_buffer vr;
struct vertex_buffer vr1;
struct vertex_buffer vr2;
struct vertex_buffer vr3;
struct vertex_buffer new_pos;
float temp0,temp1,temp2;
float alpha1;
float alpha2;
float beta;
int index;
int index1;
objective_function o;
//float temp[2][2];
float precision=0.000001;
float sn;
float sn_1;
float sn_temp;
float lamda;
float slope;


o.coeff_sqr_x=o.coeff_sqr_y=o.coeff_xy=o.rest.coeff_x=o.rest.coeff_y=o.rest.const_coeff=0;


if(row<vertices){
index1=row*max;

//if(row==1)printf("**greetings from row 60\n");
if(fan[index1+1]){
//for(int i=0;i<max;i++){
//fan_list[ty][i]=fan[row*max+i];
//if(row==60)
//printf("%d\n",fan_list[ty][i]);
//}


//if(fan_list[ty][1]==1){
//perform smoothing

count=fan[index1];

vr.x=d_vb[fan[index1+2]].x;
vr.y=d_vb[fan[index1+2]].y;
vr1.x=d_vb[row].x;
vr1.y=d_vb[row].y;
vr2.x=d_vb[fan[index1+count+1]].x;
vr2.y=d_vb[fan[index1+count+1]].y;

for(int j=2;j<count+2;j++){
      //vr.x=d_vb[fan_list[ty][j]].x;
      //vr.y=d_vb[fan_list[ty][j]].y;
      v1.x=vr1.x-vr.x;
      v1.y=vr1.y-vr.y;

      //if(row==68)
       //printf("%f %f\n",vr.x,vr.y);

      if(j-1==count)index=2;
      else index=j+1;

      vr3.x=d_vb[fan[index1+index]].x;
      vr3.y=d_vb[fan[index1+index]].y;
      //v0.x=d_vb[fan_list[ty][index]].x-vr.x;
      //v0.y=d_vb[fan_list[ty][index]].y-vr.y;


      v0.x=vr3.x-vr.x;
      v0.y=vr3.y-vr.y;


      //if(j==2)index=count+1;
      //else index=j-1;

      //v2.x=d_vb[fan_list[ty][index]].x-vr.x;
      //v2.y=d_vb[fan_list[ty][index]].y-vr.y;

       v2.x=vr2.x-vr.x;
       v2.y=vr2.y-vr.y;

      temp0=sqrtf(powf(v0.x,2)+powf(v0.y,2));
      temp1=sqrtf(powf(v1.x,2)+powf(v1.y,2));
      temp2=sqrtf(powf(v2.x,2)+powf(v2.y,2));

      alpha1=acos((v1.x*v2.x+v1.y*v2.y)/(temp1*temp2));
      alpha2=acos((v1.x*v0.x+v1.y*v0.y)/(temp1*temp0));

      beta=(alpha2+alpha1)/2;
      beta=alpha2-beta;

      
      new_pos.x=vr.x+(vr1.x-vr.x)*cos(beta)-(vr1.y-vr.y)*sin(beta);
      new_pos.y=vr.y+(vr1.x-vr.x)*sin(beta)+(vr1.y-vr.y)*cos(beta);


      temp0=new_pos.y-vr.y;
      temp1=vr.x-new_pos.x;
      temp2=vr.y*new_pos.x-vr.x*new_pos.y;
      slope=sqrtf(powf(temp0,2)+powf(temp1,2));
      temp0=funct[index1+j-2].coeff_x=temp0/slope;
      temp1=funct[index1+j-2].coeff_y=temp1/slope;
      temp2=funct[index1+j-2].const_coeff=temp2/slope;

//     if(row==104)printf("%f %f %f\n",temp0,temp1,temp2);


     o.coeff_sqr_x+=powf(temp0,2);
     o.coeff_sqr_y+=powf(temp1,2);
     o.coeff_xy+=2*temp0*temp1;
     o.rest.coeff_x+=2*temp0*temp2;
     o.rest.coeff_y+=2*temp1*temp2;
     o.rest.const_coeff+=powf(temp2,2);
      

      vr2=vr;
      vr=vr3;

   }//Inner loop ends here
   index=2*index1;
   //if(row==104)printf("%f %f %f %f %f %f\n",o.coeff_sqr_x,o.coeff_sqr_y,o.coeff_xy,o.rest.coeff_x,o.rest.coeff_y,o.rest.const_coeff);
   //temp[0][0]=temp[0][1]=temp[1][0]=temp[1][1]=0;
     vr3.x=vr3.y=vr2.x=vr2.y=0;
      for(int k=0;k<count;k++){
	temp0=jacobian[index+2*k]=funct[index1+k].coeff_x;
	temp1=jacobian[index+2*k+1]=funct[index1+k].coeff_y;
        vr2.y+=powf(temp0,2);   
        vr2.x+=(temp0*temp1);
        vr3.x+=powf(temp1,2);

       //if(row==104)printf("%f %f\n",jacobian[row*max*2+2*k],jacobian[row*max*2+2*k+1]);
   }

  // if(row==104)
   //   for(int k=0;k<count;k++){
    //   printf("%f %f\n",jacobian[row*max*2+2*k],jacobian[row*max*2+2*k+1]);
    //}

/*   temp[0][0]=temp[0][1]=temp[1][0]=temp[1][1]=0;
   for(int k=0;k<count;k++){
     temp[1][1]+=powf(jacobian[row*max*2+2*k],2);   
     temp[1][0]+=jacobian[row*max*2+2*k]*jacobian[row*max*2+2*k+1];
     temp[0][0]+=powf(jacobian[row*max*2+2*k+1],2);
    // if(row==104)
     //printf("%f %f \n",funct[row*max+k].coeff_x,funct[row*max+k].coeff_y);
   }
*/
   //temp[0][1]=temp[1][0];
     vr3.y=vr2.x;
   //  if(row==104)printf("%f %f %f %f\n",temp[0][0],temp[0][1],temp[1][0],temp[1][1]);

   //slope=(temp[0][0]*temp[1][1])-(temp[0][1]*temp[1][0]);
     slope=(vr3.x*vr2.y-vr3.y*vr2.x);

    vr3.x/=slope;
    vr3.y/=-slope;
    vr2.x/=-slope;
    vr2.y/=slope;

 //  if(row==104)printf("%f %f %f %f\n",temp[0][0],temp[0][1],temp[1][0],temp[1][1]);

   for(int ii=0;ii<count;ii++){
    d1[index+ii]=vr3.x*jacobian[index+2*ii];
    d1[index+ii]+=(vr3.y*jacobian[index+2*ii+1]);
    d1[index+ii+count]=vr2.x*jacobian[index+2*ii];
    d1[index+ii+count]+=(vr2.y*jacobian[index+2*ii+1]);

  //  if(row==104){printf("%f %f %f %f %f %f\n",temp[0][0],temp[0][1],temp[1][0],temp[1][1],jacobian[row*max*2+2*ii],jacobian[row*max*2+2*ii+1]);
    // printf("%f %f\n",d1[row*max*2+ii],d1[row*max*2+ii+count]);}
   }

//Initial guess of internal vertex
   v1.x=vr1.x;
   v1.y=vr1.y;

//Here begins the calculation of main algorithmic loop
  for(int kk=0;kk<10;kk++){
    //cout<<v1.x<<" "<<v1.y<<endl;
    //calculation of function vector reusing the jacobain matrix this time for function vector
  for(int ii=0;ii<count;ii++){
   jacobian[index+ii]=v1.x*funct[index1+ii].coeff_x+v1.y*funct[index1+ii].coeff_y+funct[index1+ii].const_coeff;
  // if(row==104)printf("****%f\n",jacobian[row*max*2+ii]);
    //printf("****%f %f \n",funct[row*max+ii].coeff_x,funct[row*max+ii].coeff_y); 
  }

   //First step of Algorithm
    sn=powf(v1.x,2)*o.coeff_sqr_x+powf(v1.y,2)*o.coeff_sqr_y+v1.x*v1.y*o.coeff_xy;
    sn+=v1.x*o.rest.coeff_x+v1.y*o.rest.coeff_y+o.rest.const_coeff;

  
  v0.x=v0.y=0;//Reinitializing the current vertex to calculate the new position
  for(int ii=0;ii<count;ii++){
    v0.x+=d1[index+ii]*jacobian[index+ii];
    v0.y+=d1[index+ii+count]*jacobian[index+ii];
  }
  
 //new coordinates i.e. xn+1 

  //if(row==104)printf("*****%f %f\n",v0.x,v0.y);
  v2.x=v1.x+v0.x;
  v2.y=v1.y+v0.y;

  sn_1=powf(v2.x,2)*o.coeff_sqr_x+powf(v2.y,2)*o.coeff_sqr_y+v2.x*v2.y*o.coeff_xy;
  sn_1+=v2.x*o.rest.coeff_x+v2.y*o.rest.coeff_y+o.rest.const_coeff;

  if(fabs((sn_1-sn)/sn) < precision){
    //puts("reached");
     break;}
   
  lamda=1;//step 3 of algorithm

 if(sn_1<sn){//puts("Here");
   v1.x=v2.x;v1.y=v2.y;continue;}//step 4 of algorithm


  //calculation of lamda using derivative;step 5 of algorithm

  lamda=(2*o.coeff_sqr_x*v0.x+o.coeff_xy*v0.y)*v1.x;
  lamda+=(2*o.coeff_sqr_y*v0.y+o.coeff_xy*v0.x)*v1.y;
  lamda+=o.rest.coeff_x*v0.x+o.rest.coeff_y*v0.y;
  lamda/=(o.coeff_sqr_x*powf(v0.x,2)+o.coeff_sqr_y*powf(v0.y,2)+o.coeff_xy*v0.x*v0.y);
  lamda/=(-2);

  v2.x=v1.x+lamda*v0.x;
  v2.y=v1.y+lamda*v0.y;

  sn_temp=powf(v2.x,2)*o.coeff_sqr_x+powf(v2.y,2)*o.coeff_sqr_y+v2.x*v2.y*o.coeff_xy;
  sn_temp+=v2.x*o.rest.coeff_x+v2.y*o.rest.coeff_y+o.rest.const_coeff; 

  //Step 6 of an algorithm
  if(sn>sn_1){
    if(sn_1<sn_temp)lamda=1;
   }
  else if(sn<sn_temp)lamda=0;

  //cout<<"lamda="<<lamda<<endl;

  //Step 7 of algorithm

   //if(row==104)printf("lamda=%f\n",lamda);
    int k=kk;

    while(k<10){
    
     if(sn_1<sn){break;}
      //puts("***************");
      lamda/=2;
      v1.x=v1.x+lamda*v0.x;
      v1.y=v1.y+lamda*v0.y;

      sn=powf(v1.x,2)*o.coeff_sqr_x+powf(v1.y,2)*o.coeff_sqr_y+v1.x*v1.y*o.coeff_xy;
      sn+=v1.x*o.rest.coeff_x+v1.y*o.rest.coeff_y+o.rest.const_coeff;

      for(int ii=0;ii<count;ii++)
       jacobian[index+ii]=v1.x*funct[index1+ii].coeff_x+v1.y*funct[index1+ii].coeff_y+funct[index1+ii].const_coeff;

      v0.x=v0.y=0;//Reinitializing the current vertex to calculate the new position
     for(int ii=0;ii<count;ii++){
       v0.x+=d1[index+ii]*jacobian[index+ii];
       v0.y+=d1[index+ii+count]*jacobian[index+ii];
     }

     v2.x=v1.x+v0.x;
     v2.y=v1.y+v0.y;

      sn_1=powf(v2.x,2)*o.coeff_sqr_x+powf(v2.y,2)*o.coeff_sqr_x+v2.x*v2.y*o.coeff_xy;
      sn_1+=v2.x*o.rest.coeff_x+v2.y*o.rest.coeff_y+o.rest.const_coeff;

      k++;
      }

    if(k==10)break;

   }//Algorithm loop ends here*/

   //if(row==104)printf("%f %f\n",v1.x,v1.y);
   result[row].x=v1.x;
   result[row].y=v1.y;
   }

else{
result[row].x=d_vb[row].x;
result[row].y=d_vb[row].y;
}

}

}

